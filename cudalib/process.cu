#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#define THREAD_NUM 512
#define BLOCK_NUM 32

hipError_t cvtWithCuda(float *c,  unsigned char *a, int size,int idx);
hipError_t clipWithCuda(float *c,  unsigned char *a, int size,int idx);
__global__ void cvtKernel(float *c,  unsigned char *a,int size)  
{        
        const int tid = threadIdx.x;
        const int bid = blockIdx.x;
        int i;
        for (i = bid * THREAD_NUM + tid; i < size; i += THREAD_NUM * BLOCK_NUM){
                    c[i] = (float)a[i]/255.0;
        }
} 

__global__ void clipKernel(float *c,  unsigned char *a,int size)  
{        
        const int tid = threadIdx.x;
        const int bid = blockIdx.x;
        int i;
        float res;
        for (i = bid * THREAD_NUM + tid; i < size; i += THREAD_NUM * BLOCK_NUM){     
            res =(float)a[i] - c[i]*255.0;
            if(res < 0)
                res = 0;
            if(res > 255)
                res = 255;
            a[i] = (unsigned char)res;
        }
} 

hipError_t cvtWithCuda(float *c, unsigned char*a, int  size,int idx)  
{  
    unsigned char *dev_a = 0; 
    float *dev_c = c;  
    hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system. 
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop,idx);
    cudaStatus = hipSetDevice(idx);  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");  
        goto Error;  
    }  

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(unsigned char));  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipMalloc failed!");  
        goto Error;  
    }  
      
    // Copy input vectors from host memory to GPU buffers.  
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(unsigned char), hipMemcpyHostToDevice);  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipMemcpy failed!");  
        goto Error;  
    }  
 
    // Launch a kernel on the GPU with one thread for each element.  
    cvtKernel<<<BLOCK_NUM, THREAD_NUM>>>(dev_c, dev_a,size);  
    cudaStatus = hipDeviceSynchronize(); 
    if(cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);  
        goto Error;  
    }  
Error:  
    hipFree(dev_a);  
      
    return cudaStatus;  
}

hipError_t clipWithCuda(float *c, unsigned char*a,int size,int idx)  
{  
    unsigned char *dev_a = 0;  
    float *dev_c = c;  
    hipError_t cudaStatus; 
    // Choose which GPU to run on, change this on a multi-GPU system.  
    cudaStatus = hipSetDevice(idx);  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");  
        goto Error;  
    }  
  
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(unsigned char));  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipMalloc failed!");  
        goto Error;  
    }  
    // Copy input vectors from host memory to GPU buffers.  
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(unsigned char), hipMemcpyHostToDevice);  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipMemcpy failed!");  
        goto Error;  
    }  
 
    // Launch a kernel on the GPU with one thread for each element.  
    clipKernel<<<BLOCK_NUM, THREAD_NUM>>>(dev_c, dev_a,size);  
    cudaStatus = hipDeviceSynchronize(); 
    if(cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching clipKernel!\n", cudaStatus);  
        goto Error;  
    }  
  
    // Copy output vector from GPU buffer to host memory.  
    cudaStatus = hipMemcpy(a, dev_a, size * sizeof(unsigned char), hipMemcpyDeviceToHost);  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipMemcpy failed!");  
        goto Error;  
    }  

Error:  
    hipFree(dev_a);  
      
    return cudaStatus; 
}

extern "C" int Cudacvt(float *c,unsigned char *a,int w,int h,int idx){
    hipError_t cudaStatus;  
    cudaStatus = cvtWithCuda(c, a,w*h,idx);  
    if (cudaStatus != hipSuccess)   
    {  
        fprintf(stderr, "addWithCuda failed!");  
        return -1;  
    }  
    return 0;
}
extern "C" int Cudaclip(float *c,unsigned char *a,int w,int h,int idx){
    // Add vectors in parallel.  
    hipError_t cudaStatus;  
    cudaStatus = clipWithCuda(c,a,w*h,idx);  
    if (cudaStatus != hipSuccess)   
    {  
        fprintf(stderr, "addWithCuda failed!");  
        return -1;  
    }  
    return 0;
}
